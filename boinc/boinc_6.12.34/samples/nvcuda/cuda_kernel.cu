#include "hip/hip_runtime.h"
// This file is part of BOINC.
// http://boinc.berkeley.edu
// Copyright (C) 2008 University of California
//
// BOINC is free software; you can redistribute it and/or modify it
// under the terms of the GNU Lesser General Public License
// as published by the Free Software Foundation,
// either version 3 of the License, or (at your option) any later version.
//
// BOINC is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
// See the GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with BOINC.  If not, see <http://www.gnu.org/licenses/>.
//
// This file contains kernel definition for matrix inversion. The external function
// "invert" serves as an interface between cuda_kernel.cu and cuda.cpp
//
// See http://boinc.berkeley.edu/trac/wiki/GPUApp for any compiling issues
// Contributor: Tuan Le (tuanle86@berkeley.edu)

// When VERIFY is defined, the sum of squared errors is calculated between the
// identity matrix and the product A * incerse(A). For debugging...
//#define VERIFY 1
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "cuda_config.h"

__global__ void GEStep1A(REAL * AI, int i, int n2, int lda2) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k>i && k < n2 && AI[i*lda2+k]!=0) {
        REAL multiplyer = -AI[i*lda2+k]/AI[i*lda2+i];
        int n = n2 / 2;
        for (int j = i+1; j < n; j++) {
            AI[j*lda2+k] += multiplyer*AI[j*lda2+i];
        }
    }
}

__global__ void GEStep2(REAL * AI,REAL diag,int i, int n2, int lda2) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k < n2) {
        AI[i*lda2+k] /= diag;
    }
}

__global__ void GEStep3(REAL * AI,int i, int n2, int lda2) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k > i && k < n2) {
        REAL multiplyer = -AI[i*lda2+k];
        for (int j = 0; j < i; j++) {
            AI[j*lda2+k] += multiplyer*AI[j*lda2+i];
        }
    }
}

/* Helper function for invert. Kernel calls are made in this function */
void invertge(REAL * AI_d, int lda, int n) {
    int lda2 = lda * 2;
    // perform elementary row operations till A in AI becomes identity matrix
    for (int i = 0; i < n; i++) {
        GEStep1A<<<(int)ceil((float)(1+(2*n-1)/32)),32>>>(AI_d,i,n*2, lda2);
        CUDACHECK;
        hipDeviceSynchronize();
    }

    for (int i = n-1; i >= 0; i--) {
        REAL diag = 1.0;
        SAFECALL(hipMemcpy(&diag, &AI_d[i*lda2+i], sizeof(REAL), hipMemcpyDeviceToHost));
        GEStep2<<<(int)ceil((float)(1+(n*2-1)/32)),32>>>(AI_d,diag,i,n*2, lda2);
        CUDACHECK;

        GEStep3<<<(int)ceil((float)(1+(n*2-1)/32)),32>>>(AI_d,i,n*2, lda2);
        CUDACHECK;
        hipDeviceSynchronize();
        CUDACHECK;
    }
}

/* inverts nxn matrix A and stores result back in A */
extern void invert(REAL * A, int n) {
    fprintf(stderr,"starting inversion n = %d ", n);
    volatile clock_t gputime;
    gputime=clock();

    int lda = ((n+15)&~15|16);
    REAL * AI = (REAL *)malloc(sizeof(REAL)*(n*lda*2));
    memset(AI,0,sizeof(REAL)*n*lda*2);
    for (int i = 0; i < n; i++) {
        memcpy(&AI[lda*i*2], &A[n*i], sizeof(REAL)*n);
        AI[lda*i*2+n+i] = 1;
    }

    REAL * AI_d;
    SAFECALL(hipMalloc((void **) &AI_d, sizeof(REAL)*n*lda*2));
    SAFECALL(hipMemcpy(AI_d, AI, sizeof(REAL)*n*lda*2, hipMemcpyHostToDevice));

    invertge(AI_d, lda, n);
    SAFECALL(hipMemcpy(AI, AI_d, sizeof(REAL)*n*lda*2, hipMemcpyDeviceToHost));
    hipFree(AI_d);
    gputime=clock()-gputime;fprintf(stderr, " %7.1f ms ",gputime/1.e3f);
    fprintf(stderr, " %7.2f Gflops", 1e-3*(3.0)*n*n*n/3.0/gputime);

#ifdef VERIFY	
	// let's verify that
    REAL error=0.0;
    // multiply inverse*xcopy, should be Identity matrix
    for (int k = 0; k < n; k++) {
        for (int j = 0; j < n; j++) {
            REAL sum = 0;
            for (int i = 0; i < n; i++) {
                sum += AI[j*lda*2+n+i]*A[i*n+k];
            }
            if (j!=k) {
                error += sum * sum;
            } else {
                error += (1.0-sum) * (1.0-sum);
            }
        }
    }
    fprintf(stderr, " %6.2f SSE", error);
#endif	

    for (int i = 0; i < n; i++) {
        memcpy(&A[n*i], &AI[lda*i*2+n], sizeof(REAL)*n);
    }
    free(AI);
    fprintf(stderr," done!\n");
}